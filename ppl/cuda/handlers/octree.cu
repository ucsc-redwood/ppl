#pragma once

#include <spdlog/spdlog.h>

#include "cuda/helper.cuh"
#include "handlers/octree.h"

explicit OctreeHandler::OctreeHandler(const size_t n_octr_nodes_to_allocate)
    : n_octr_nodes_to_allocate(n_octr_nodes_to_allocate), n_oct_nodes() {
  // CHECK_CUDA_CALL(
  //     hipMallocManaged(reinterpret_cast<void**>(&u_children),
  //                       n_octr_nodes_to_allocate * 8 * sizeof(int)));

  MALLOC_MANAGED(&u_children, n_octr_nodes_to_allocate * 8);
  MALLOC_MANAGED(&u_corner, n_octr_nodes_to_allocate);
  MALLOC_MANAGED(&u_cell_size, n_octr_nodes_to_allocate);
  MALLOC_MANAGED(&u_child_node_mask, n_octr_nodes_to_allocate);
  MALLOC_MANAGED(&u_child_leaf_mask, n_octr_nodes_to_allocate);
  SYNC_DEVICE();

  spdlog::trace("On constructor: OctreeHandler, n: {}",
                n_octr_nodes_to_allocate);
}

~OctreeHandler::OctreeHandler() {
  CUDA_FREE(u_children);
  CUDA_FREE(u_corner);
  CUDA_FREE(u_cell_size);
  CUDA_FREE(u_child_node_mask);
  CUDA_FREE(u_child_leaf_mask);

  spdlog::trace("On destructor: OctreeHandler");
}

// void attachStreamSingle(const hipStream_t stream) const {
//   ATTACH_STREAM_SINGLE(u_children);
//   ATTACH_STREAM_SINGLE(u_corner);
//   ATTACH_STREAM_SINGLE(u_cell_size);
//   ATTACH_STREAM_SINGLE(u_child_node_mask);
//   ATTACH_STREAM_SINGLE(u_child_leaf_mask);
// }

// void attachStreamGlobal(const hipStream_t stream) const {
//   ATTACH_STREAM_GLOBAL(u_children);
//   ATTACH_STREAM_GLOBAL(u_corner);
//   ATTACH_STREAM_GLOBAL(u_cell_size);
//   ATTACH_STREAM_GLOBAL(u_child_node_mask);
//   ATTACH_STREAM_GLOBAL(u_child_leaf_mask);
// }

// void attachStreamHost(const hipStream_t stream) const {
//   ATTACH_STREAM_HOST(u_children);
//   ATTACH_STREAM_HOST(u_corner);
//   ATTACH_STREAM_HOST(u_cell_size);
//   ATTACH_STREAM_HOST(u_child_node_mask);
//   ATTACH_STREAM_HOST(u_child_leaf_mask);
//   SYNC_STREAM(stream);
// }
