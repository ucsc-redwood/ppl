#include "hip/hip_runtime.h"
/******************************************************************************
 * OneSweep
 *
 * Author:  Thomas Smith 9/13/2023
 *
 * Based off of Research by:
 *          Andy Adinets, Nvidia Corporation
 *          Duane Merrill, Nvidia Corporation
 *          https://research.nvidia.com/publication/2022-06_onesweep-faster-least-significant-digit-radix-sort-gpus
 *
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/util_math.cuh>

#include "cuda/kernels/common.cuh"

namespace gpu {
#define RADIX 256       // Number of digit bins
#define RADIX_MASK 255  // Mask of digit bins, to extract digits
#define RADIX_LOG 8     // log2(RADIX)

#define SEC_RADIX_START 256
#define THIRD_RADIX_START 512
#define FOURTH_RADIX_START 768

// For the upfront global histogram kernel
#define G_HIST_PART_SIZE 65536
#define G_HIST_VEC_SIZE 16384

// For the digit binning
#define BIN_PART_SIZE 7680  // Partition tile size in k_DigitBinning

#define BIN_HISTS_SIZE \
  4096  // Total size of warp histograms in shared memory in k_DigitBinning

#define BIN_SUB_PART_SIZE \
  480  // Subpartition tile size of a single warp in k_DigitBinning

#define BIN_WARPS 16  // Warps per threadblock in k_DigitBinning

#define BIN_KEYS_PER_THREAD 15  // Keys per thread in k_DigitBinning

#define BIN_SUB_PART_START \
  (WARP_INDEX * BIN_SUB_PART_SIZE)  // Starting offset of a subpartition tile

#define BIN_PART_START \
  (partitionIndex * BIN_PART_SIZE)  // Starting offset of a partition tile

// for the chained scan with decoupled lookback
#define FLAG_NOT_READY \
  0  // Flag value inidicating neither inclusive sum, nor reduction of a
     // partition tile is ready

#define FLAG_REDUCTION \
  1  // Flag value indicating reduction of a partition tile is ready

#define FLAG_INCLUSIVE \
  2  // Flag value indicating inclusive sum of a partition tile is ready

#define FLAG_MASK 3

__global__ void k_GlobalHistogram(unsigned int* sort,
                                  unsigned int* global_histogram,
                                  const unsigned int size) {
  __shared__ unsigned int s_globalHistFirst[RADIX * 2];
  __shared__ unsigned int s_globalHistSec[RADIX * 2];
  __shared__ unsigned int s_globalHistThird[RADIX * 2];
  __shared__ unsigned int s_globalHistFourth[RADIX * 2];

  const auto logicalBlocks = hipcub::DivideAndRoundUp(size, G_HIST_PART_SIZE);

  for (auto yanwen_block_id = blockIdx.x; yanwen_block_id < logicalBlocks;
       yanwen_block_id += gridDim.x) {
    // clear shared memory
    for (unsigned int i = threadIdx.x; i < RADIX * 2; i += blockDim.x) {
      s_globalHistFirst[i] = 0;
      s_globalHistSec[i] = 0;
      s_globalHistThird[i] = 0;
      s_globalHistFourth[i] = 0;
    }
    __syncthreads();

    // histogram
    {
      // 64 threads : 1 histogram in shared memory
      unsigned int* s_wavesHistFirst =
          &s_globalHistFirst[threadIdx.x / 64 * RADIX];
      unsigned int* s_wavesHistSec = &s_globalHistSec[threadIdx.x / 64 * RADIX];
      unsigned int* s_wavesHistThird =
          &s_globalHistThird[threadIdx.x / 64 * RADIX];
      unsigned int* s_wavesHistFourth =
          &s_globalHistFourth[threadIdx.x / 64 * RADIX];

      if (yanwen_block_id < logicalBlocks - 1) {
        const unsigned int partEnd = (yanwen_block_id + 1) * G_HIST_VEC_SIZE;
        for (unsigned int i = threadIdx.x + (yanwen_block_id * G_HIST_VEC_SIZE);
             i < partEnd;
             i += blockDim.x) {
          uint4 t[1] = {reinterpret_cast<uint4*>(sort)[i]};

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t*>(t)[0]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t*>(t)[1]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t*>(t)[2]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t*>(t)[3]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t*>(t)[4]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t*>(t)[5]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t*>(t)[6]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t*>(t)[7]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t*>(t)[8]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t*>(t)[9]], 1);
          atomicAdd(&s_globalHistThird[reinterpret_cast<uint8_t*>(t)[10]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t*>(t)[11]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t*>(t)[12]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t*>(t)[13]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t*>(t)[14]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t*>(t)[15]], 1);
        }
      }

      if (yanwen_block_id == logicalBlocks - 1) {
        for (unsigned int i =
                 threadIdx.x + (yanwen_block_id * G_HIST_PART_SIZE);
             i < size;
             i += blockDim.x) {
          unsigned int t[1] = {sort[i]};
          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t*>(t)[0]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t*>(t)[1]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t*>(t)[2]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t*>(t)[3]], 1);
        }
      }
    }
    __syncthreads();

    // reduce and add to device
    for (unsigned int i = threadIdx.x; i < RADIX; i += blockDim.x) {
      atomicAdd(&global_histogram[i],
                s_globalHistFirst[i] + s_globalHistFirst[i + RADIX]);
      atomicAdd(&global_histogram[i + SEC_RADIX_START],
                s_globalHistSec[i] + s_globalHistSec[i + RADIX]);
      atomicAdd(&global_histogram[i + THIRD_RADIX_START],
                s_globalHistThird[i] + s_globalHistThird[i + RADIX]);
      atomicAdd(&global_histogram[i + FOURTH_RADIX_START],
                s_globalHistFourth[i] + s_globalHistFourth[i + RADIX]);
    }
  }
}

// fixed to use 4 blocks, and 'radix' (256) threads
__global__ void k_Scan(unsigned int* globalHistogram,
                       unsigned int* firstPassHistogram,
                       unsigned int* secPassHistogram,
                       unsigned int* thirdPassHistogram,
                       unsigned int* fourthPassHistogram) {
  __shared__ unsigned int s_scan[RADIX];

  s_scan[threadIdx.x] = InclusiveWarpScanCircularShift(
      globalHistogram[threadIdx.x + blockIdx.x * RADIX]);
  __syncthreads();

  if (threadIdx.x < (RADIX >> LANE_LOG))
    s_scan[threadIdx.x << LANE_LOG] =
        ActiveExclusiveWarpScan(s_scan[threadIdx.x << LANE_LOG]);
  __syncthreads();

  switch (blockIdx.x) {
    case 0:
      firstPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 1:
      secPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 2:
      thirdPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 3:
      fourthPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    default:
      break;
  }
}

__global__ void k_DigitBinningPass_Original(
    unsigned int* sort,
    unsigned int* alt,
    volatile unsigned int* passHistogram,
    volatile unsigned int* index,
    unsigned int size,
    unsigned int radixShift) {
  __shared__ unsigned int s_warpHistograms[BIN_PART_SIZE];
  __shared__ unsigned int s_localHistogram[RADIX];
  volatile unsigned int* s_warpHist =
      &s_warpHistograms[WARP_INDEX << RADIX_LOG];

  // clear shared memory
  for (unsigned int i = threadIdx.x; i < BIN_HISTS_SIZE;
       i += blockDim.x)  // unnecessary work for last partion but still a win to
                         // avoid another barrier
    s_warpHistograms[i] = 0;

  // atomically assign partition tiles
  if (threadIdx.x == 0)
    s_warpHistograms[BIN_PART_SIZE - 1] =
        atomicAdd((unsigned int*)&index[radixShift >> 3], 1);
  __syncthreads();
  const unsigned int partitionIndex = s_warpHistograms[BIN_PART_SIZE - 1];

  // To handle input sizes not perfect multiples of the partition tile size
  if (partitionIndex < gridDim.x - 1) {
    // load keys
    unsigned int keys[BIN_KEYS_PER_THREAD];
#pragma unroll
    for (unsigned int i = 0,
                      t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START;
         i < BIN_KEYS_PER_THREAD;
         ++i, t += LANE_COUNT)
      keys[i] = sort[t];

    uint16_t offsets[BIN_KEYS_PER_THREAD];

// WLMS
#pragma unroll
    for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
      // CUB version "match any"
      /*
      unsigned warpFlags;
      #pragma unroll
      for (int k = 0; k < RADIX_LOG; ++k)
      {
          unsigned int mask;
          unsigned int current_bit = 1 << k + radixShift;
          asm("{\n"
              "    .reg .pred p;\n"
              "    and.b32 %0, %1, %2;"
              "    setp.ne.u32 p, %0, 0;\n"
              "    vote.ballot.sync.b32 %0, p, 0xffffffff;\n"
              "    @!p not.b32 %0, %0;\n"
              "}\n" : "=r"(mask) : "r"(keys[i]), "r"(current_bit));
          warpFlags = (k == 0) ? mask : warpFlags & mask;
      }
      const unsigned int bits = __popc(warpFlags & getLaneMaskLt());
      */
      unsigned warpFlags = 0xffffffff;
#pragma unroll
      for (int k = 0; k < RADIX_LOG; ++k) {
        const bool t2 = keys[i] >> k + radixShift & 1;
        warpFlags &= (t2 ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t2);
      }
      const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

      // An alternative, but slightly slower version.
      /*
      offsets[i] = s_warpHist[keys[i] >> radixShift & RADIX_MASK] + bits;
      __syncwarp(0xffffffff);
      if (bits == 0)
          s_warpHist[keys[i] >> radixShift & RADIX_MASK] += __popc(warpFlags);
      __syncwarp(0xffffffff);
      */
      unsigned int preIncrementVal;
      if (bits == 0)
        preIncrementVal = atomicAdd(
            (unsigned int*)&s_warpHist[keys[i] >> radixShift & RADIX_MASK],
            __popc(warpFlags));

      offsets[i] =
          __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) + bits;
    }
    __syncthreads();

    // exclusive prefix sum up the warp histograms
    if (threadIdx.x < RADIX) {
      unsigned int reduction = s_warpHistograms[threadIdx.x];
      for (unsigned int i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE;
           i += RADIX) {
        reduction += s_warpHistograms[i];
        s_warpHistograms[i] = reduction - s_warpHistograms[i];
      }

      atomicAdd((unsigned int*)&passHistogram[threadIdx.x +
                                              (partitionIndex + 1) * RADIX],
                FLAG_REDUCTION | reduction << 2);

      // begin the exclusive prefix sum across the reductions
      s_localHistogram[threadIdx.x] = InclusiveWarpScanCircularShift(reduction);
    }
    __syncthreads();

    if (threadIdx.x < (RADIX >> LANE_LOG))
      s_localHistogram[threadIdx.x << LANE_LOG] =
          ActiveExclusiveWarpScan(s_localHistogram[threadIdx.x << LANE_LOG]);
    __syncthreads();

    if (threadIdx.x < RADIX && getLaneId())
      s_localHistogram[threadIdx.x] +=
          __shfl_sync(0xfffffffe, s_localHistogram[threadIdx.x - 1], 1);
    __syncthreads();

    // update offsets
    if (WARP_INDEX) {
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
        const unsigned int t2 = keys[i] >> radixShift & RADIX_MASK;
        offsets[i] += s_warpHist[t2] + s_localHistogram[t2];
      }
    } else {
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        offsets[i] += s_localHistogram[keys[i] >> radixShift & RADIX_MASK];
    }
    __syncthreads();

// scatter keys into shared memory
#pragma unroll
    for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
      s_warpHistograms[offsets[i]] = keys[i];

    // split the warps into single thread cooperative groups and lookback
    if (threadIdx.x < RADIX) {
      unsigned int reduction = 0;
      for (unsigned int k = partitionIndex; k >= 0;) {
        const unsigned int flagPayload = passHistogram[threadIdx.x + k * RADIX];

        if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
          reduction += flagPayload >> 2;
          atomicAdd((unsigned int*)&passHistogram[threadIdx.x +
                                                  (partitionIndex + 1) * RADIX],
                    1 | (reduction << 2));
          s_localHistogram[threadIdx.x] =
              reduction - s_localHistogram[threadIdx.x];
          break;
        }

        if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
          reduction += flagPayload >> 2;
          k--;
        }
      }
    }
    __syncthreads();

// scatter runs of keys into device memory
#pragma unroll
    for (unsigned int i = threadIdx.x; i < BIN_PART_SIZE; i += blockDim.x)
      alt[s_localHistogram[s_warpHistograms[i] >> radixShift & RADIX_MASK] +
          i] = s_warpHistograms[i];
  }

  // Process the final partition slightly differently
  if (partitionIndex == gridDim.x - 1) {
    // immediately begin lookback
    if (threadIdx.x < RADIX) {
      if (partitionIndex) {
        unsigned int reduction = 0;
        for (unsigned int k = partitionIndex; k >= 0;) {
          const unsigned int flagPayload =
              passHistogram[threadIdx.x + k * RADIX];

          if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
            reduction += flagPayload >> 2;
            s_localHistogram[threadIdx.x] = reduction;
            break;
          }

          if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
            reduction += flagPayload >> 2;
            k--;
          }
        }
      } else {
        s_localHistogram[threadIdx.x] = passHistogram[threadIdx.x] >> 2;
      }
    }
    __syncthreads();

    const unsigned int partEnd = BIN_PART_START + BIN_PART_SIZE;
    for (unsigned int i = threadIdx.x + BIN_PART_START; i < partEnd;
         i += blockDim.x) {
      unsigned int key;
      unsigned int offset;
      unsigned warpFlags = 0xffffffff;

      if (i < size) key = sort[i];

#pragma unroll
      for (unsigned int k = 0; k < RADIX_LOG; ++k) {
        const bool t = key >> k + radixShift & 1;
        warpFlags &= (t ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t);
      }
      const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

#pragma unroll
      for (unsigned int k = 0; k < BIN_WARPS; ++k) {
        unsigned int preIncrementVal;
        if (WARP_INDEX == k && bits == 0 && i < size)
          preIncrementVal =
              atomicAdd(&s_localHistogram[key >> radixShift & RADIX_MASK],
                        __popc(warpFlags));

        if (WARP_INDEX == k)
          offset =
              __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) +
              bits;
        __syncthreads();
      }

      if (i < size) alt[offset] = key;
    }
  }
}

//
// ============================================================================
// Yanwen's version
//
// ============================================================================

__global__ void k_DigitBinningPass(unsigned int* sort,
                                   unsigned int* alt,
                                   volatile unsigned int* passHistogram,
                                   volatile unsigned int* index,
                                   unsigned int size,
                                   unsigned int radixShift) {
  __shared__ unsigned int s_warpHistograms[BIN_PART_SIZE];
  __shared__ unsigned int s_localHistogram[RADIX];

  volatile unsigned int* s_warpHist =
      &s_warpHistograms[WARP_INDEX << RADIX_LOG];

  const auto logicalBlocks = hipcub::DivideAndRoundUp(size, BIN_PART_SIZE);

  for (auto yanwen_block_id = blockIdx.x; yanwen_block_id < logicalBlocks;
       yanwen_block_id += gridDim.x) {
    // clear shared memory
    for (unsigned int i = threadIdx.x; i < BIN_HISTS_SIZE;
         i += blockDim.x)  // unnecessary work for last partion but still a win
                           // to avoid another barrier
      s_warpHistograms[i] = 0;

    // atomically assign partition tiles
    if (threadIdx.x == 0)
      s_warpHistograms[BIN_PART_SIZE - 1] =
          atomicAdd((unsigned int*)&index[radixShift >> 3], 1);
    __syncthreads();
    const unsigned int partitionIndex = s_warpHistograms[BIN_PART_SIZE - 1];

    // To handle input sizes not perfect multiples of the partition tile size
    if (partitionIndex < logicalBlocks - 1) {
      // load keys
      unsigned int keys[BIN_KEYS_PER_THREAD];
#pragma unroll
      for (unsigned int i = 0,
                        t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START;
           i < BIN_KEYS_PER_THREAD;
           ++i, t += LANE_COUNT)
        keys[i] = sort[t];

      uint16_t offsets[BIN_KEYS_PER_THREAD];

// WLMS
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
        // CUB version "match any"
        /*
        unsigned warpFlags;
        #pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k)
        {
            unsigned int mask;
            unsigned int current_bit = 1 << k + radixShift;
            asm("{\n"
                "    .reg .pred p;\n"
                "    and.b32 %0, %1, %2;"
                "    setp.ne.u32 p, %0, 0;\n"
                "    vote.ballot.sync.b32 %0, p, 0xffffffff;\n"
                "    @!p not.b32 %0, %0;\n"
                "}\n" : "=r"(mask) : "r"(keys[i]), "r"(current_bit));
            warpFlags = (k == 0) ? mask : warpFlags & mask;
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());
        */
        unsigned warpFlags = 0xffffffff;
#pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k) {
          const bool t2 = keys[i] >> k + radixShift & 1;
          warpFlags &= (t2 ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t2);
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

        // An alternative, but slightly slower version.
        /*
        offsets[i] = s_warpHist[keys[i] >> radixShift & RADIX_MASK] + bits;
        __syncwarp(0xffffffff);
        if (bits == 0)
            s_warpHist[keys[i] >> radixShift & RADIX_MASK] += __popc(warpFlags);
        __syncwarp(0xffffffff);
        */
        unsigned int preIncrementVal;
        if (bits == 0)
          preIncrementVal = atomicAdd(
              (unsigned int*)&s_warpHist[keys[i] >> radixShift & RADIX_MASK],
              __popc(warpFlags));

        offsets[i] =
            __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) +
            bits;
      }
      __syncthreads();

      // exclusive prefix sum up the warp histograms
      if (threadIdx.x < RADIX) {
        unsigned int reduction = s_warpHistograms[threadIdx.x];
        for (unsigned int i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE;
             i += RADIX) {
          reduction += s_warpHistograms[i];
          s_warpHistograms[i] = reduction - s_warpHistograms[i];
        }

        atomicAdd((unsigned int*)&passHistogram[threadIdx.x +
                                                (partitionIndex + 1) * RADIX],
                  FLAG_REDUCTION | reduction << 2);

        // begin the exclusive prefix sum across the reductions
        s_localHistogram[threadIdx.x] =
            InclusiveWarpScanCircularShift(reduction);
      }
      __syncthreads();

      if (threadIdx.x < (RADIX >> LANE_LOG))
        s_localHistogram[threadIdx.x << LANE_LOG] =
            ActiveExclusiveWarpScan(s_localHistogram[threadIdx.x << LANE_LOG]);
      __syncthreads();

      if (threadIdx.x < RADIX && getLaneId())
        s_localHistogram[threadIdx.x] +=
            __shfl_sync(0xfffffffe, s_localHistogram[threadIdx.x - 1], 1);
      __syncthreads();

      // update offsets
      if (WARP_INDEX) {
#pragma unroll
        for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
          const unsigned int t2 = keys[i] >> radixShift & RADIX_MASK;
          offsets[i] += s_warpHist[t2] + s_localHistogram[t2];
        }
      } else {
#pragma unroll
        for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
          offsets[i] += s_localHistogram[keys[i] >> radixShift & RADIX_MASK];
      }
      __syncthreads();

// scatter keys into shared memory
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        s_warpHistograms[offsets[i]] = keys[i];

      // split the warps into single thread cooperative groups and lookback
      if (threadIdx.x < RADIX) {
        unsigned int reduction = 0;
        for (unsigned int k = partitionIndex; k >= 0;) {
          const unsigned int flagPayload =
              passHistogram[threadIdx.x + k * RADIX];

          if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
            reduction += flagPayload >> 2;
            atomicAdd(
                (unsigned int*)&passHistogram[threadIdx.x +
                                              (partitionIndex + 1) * RADIX],
                1 | (reduction << 2));
            s_localHistogram[threadIdx.x] =
                reduction - s_localHistogram[threadIdx.x];
            break;
          }

          if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
            reduction += flagPayload >> 2;
            k--;
          }
        }
      }
      __syncthreads();

// scatter runs of keys into device memory
#pragma unroll
      for (unsigned int i = threadIdx.x; i < BIN_PART_SIZE; i += blockDim.x)
        alt[s_localHistogram[s_warpHistograms[i] >> radixShift & RADIX_MASK] +
            i] = s_warpHistograms[i];
    }

    // Process the final partition slightly differently
    if (partitionIndex == logicalBlocks - 1) {
      // immediately begin lookback
      if (threadIdx.x < RADIX) {
        if (partitionIndex) {
          unsigned int reduction = 0;
          for (unsigned int k = partitionIndex; k >= 0;) {
            const unsigned int flagPayload =
                passHistogram[threadIdx.x + k * RADIX];

            if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
              reduction += flagPayload >> 2;
              s_localHistogram[threadIdx.x] = reduction;
              break;
            }

            if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
              reduction += flagPayload >> 2;
              k--;
            }
          }
        } else {
          s_localHistogram[threadIdx.x] = passHistogram[threadIdx.x] >> 2;
        }
      }
      __syncthreads();

      const unsigned int partEnd = BIN_PART_START + BIN_PART_SIZE;
      for (unsigned int i = threadIdx.x + BIN_PART_START; i < partEnd;
           i += blockDim.x) {
        unsigned int key;
        unsigned int offset;
        unsigned warpFlags = 0xffffffff;

        if (i < size) key = sort[i];

#pragma unroll
        for (unsigned int k = 0; k < RADIX_LOG; ++k) {
          const bool t = key >> k + radixShift & 1;
          warpFlags &= (t ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t);
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

#pragma unroll
        for (unsigned int k = 0; k < BIN_WARPS; ++k) {
          unsigned int preIncrementVal;
          if (WARP_INDEX == k && bits == 0 && i < size)
            preIncrementVal =
                atomicAdd(&s_localHistogram[key >> radixShift & RADIX_MASK],
                          __popc(warpFlags));

          if (WARP_INDEX == k)
            offset =
                __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) +
                bits;
          __syncthreads();
        }

        if (i < size) alt[offset] = key;
      }
    }
  }
}

}  // namespace gpu
