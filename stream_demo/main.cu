#include "hip/hip_runtime.h"
#include <spdlog/spdlog.h>

#include <algorithm>
#include <array>
#include <glm/glm.hpp>

#include "cuda/helper.cuh"
#include "cuda/kernels/01_morton.cuh"

__global__ void k_DoSomethingA(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(sqrt(u_input[j].x));
  }
}

__global__ void k_DoSomethingB(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = u_input[j].x * u_input[j].y * u_input[j].z * u_input[j].w;
  }
}

__global__ void k_DoSomethingC(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(exp(u_input[j].z));
  }
}

struct Task {
  void allocate(const int n) {
    this->n = n;
    CHECK_CUDA_CALL(hipMallocManaged(&u_input, n * sizeof(glm::vec4)));
    CHECK_CUDA_CALL(hipMallocManaged(&u_output, n * sizeof(unsigned int)));

    std::generate(u_input, u_input + n, []() {
      return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
    });
  }

  ~Task() {
    CUDA_FREE(u_input);
    CUDA_FREE(u_output);
  }

  int n;
  glm::vec4* u_input;
  unsigned int* u_output;
};

void execute(Task& t, hipStream_t* stream, const int tid) {
  CHECK_CUDA_CALL(
      hipStreamAttachMemAsync(stream[tid], t.u_input, 0, hipMemAttachSingle));

  if (tid == 0) {
    k_DoSomethingA<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 1) {
    k_DoSomethingB<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 2) {
    k_DoSomethingC<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else {
    gpu::k_ComputeMortonCode<<<1, 512, 0, stream[tid]>>>(
        t.u_input, t.u_output, t.n, 0.0f, 100.0f);
  }
}

int main() {
  constexpr auto n = 1 << 20;  // 1M elements

  constexpr auto n_tasks = 100;

  // volatile auto h_original_input = new glm::vec4[n];

  glm::vec4* u_original_input;
  CHECK_CUDA_CALL(hipMallocManaged(&u_original_input, n * sizeof(glm::vec4)));

  std::generate(u_original_input, u_original_input + n, []() {
    return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
  });

  constexpr auto n_streams = 4;
  std::array<hipStream_t, n_streams> streams;

  // std::vector<Task> tasks(n_tasks);  // to dos
  // for (auto& task : tasks) {
  //   task.allocate(n);
  // }

  std::array<Task, n_streams> tasks{Task(), Task(), Task(), Task()};

  for (auto& task : tasks) {
    task.allocate(n);
  }

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  constexpr auto min_coord = 0.0f;
  constexpr auto range = 100.0f;

  hipEvent_t start, stop;
  CHECK_CUDA_CALL(hipEventCreate(&start));
  CHECK_CUDA_CALL(hipEventCreate(&stop));

  CHECK_CUDA_CALL(hipEventRecord(start, nullptr));

  // ------------------------------

  for (auto i = 0; i < n_tasks; ++i) {
    const auto my_id = i % n_streams;
    CHECK_CUDA_CALL(hipMemcpyAsync(tasks[my_id].u_input,
                                    u_original_input,
                                    n * sizeof(glm::vec4),
                                    hipMemcpyDefault,
                                    streams[my_id]));

    execute(tasks[my_id], streams.data(), my_id);
  }

  // ------------------------------

  SYNC_DEVICE();

  CHECK_CUDA_CALL(hipEventRecord(stop, nullptr));
  CHECK_CUDA_CALL(hipEventSynchronize(stop));

  float milliseconds = 0;
  CHECK_CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  spdlog::info("Total time: {} ms", milliseconds);

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }

  CHECK_CUDA_CALL(hipEventDestroy(start));
  CHECK_CUDA_CALL(hipEventDestroy(stop));

  // delete[] h_original_input;

  CUDA_FREE(u_original_input);

  return 0;
}