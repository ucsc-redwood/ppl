#include "hip/hip_runtime.h"
#include <spdlog/spdlog.h>

#include <algorithm>
#include <array>
#include <glm/glm.hpp>

#include "cuda/helper.cuh"
#include "cuda/kernels/01_morton.cuh"

__global__ void k_DoSomethingA(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(sqrt(u_input[j].x));
  }
}

__global__ void k_DoSomethingB(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(pow(u_input[j].y, 6.66));
  }
}

__global__ void k_DoSomethingC(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(exp(u_input[j].z));
  }
}

struct Task {
  void allocate(const int n) {
    this->n = n;
    MALLOC_MANAGED(&u_input, n);
    MALLOC_MANAGED(&u_output, n);

    std::generate(u_input, u_input + n, []() {
      return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
    });
  }

  ~Task() {
    CUDA_FREE(u_input);
    CUDA_FREE(u_output);
  }

  int n;
  glm::vec4* u_input;
  unsigned int* u_output;
};

void execute(Task& t, hipStream_t* stream, const int tid) {
  CHECK_CUDA_CALL(
      hipStreamAttachMemAsync(stream[tid], t.u_input, 0, hipMemAttachSingle));

  if (tid == 0) {
    k_DoSomethingA<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 1) {
    k_DoSomethingB<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 2) {
    k_DoSomethingC<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else {
    gpu::k_ComputeMortonCode<<<1, 512, 0, stream[tid]>>>(
        t.u_input, t.u_output, t.n, 0.0f, 100.0f);
  }
}

int main() {
  constexpr auto n = 1 << 20;  // 1M elements

  constexpr auto n_tasks = 100;

  constexpr auto n_streams = 4;
  std::array<hipStream_t, n_streams> streams;

  //   std::array<Task, n_streams> tasks{Task(n), Task(n), Task(n), Task(n)};

  std::vector<Task> tasks(n_tasks);  // to dos
  for (auto& task : tasks) {
    task.allocate(n);
  }

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  constexpr auto min_coord = 0.0f;
  constexpr auto range = 100.0f;

  hipEvent_t start, stop;
  CHECK_CUDA_CALL(hipEventCreate(&start));
  CHECK_CUDA_CALL(hipEventCreate(&stop));

  CHECK_CUDA_CALL(hipEventRecord(start, nullptr));

  // ------------------------------

  for (auto i = 0; i < n_tasks; ++i) {
    execute(tasks[i], streams.data(), i % n_streams);
  }

  // ------------------------------

  SYNC_DEVICE();

  CHECK_CUDA_CALL(hipEventRecord(stop, nullptr));
  CHECK_CUDA_CALL(hipEventSynchronize(stop));

  float milliseconds = 0;
  CHECK_CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  spdlog::info("Total time: {} ms", milliseconds);

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }

  CHECK_CUDA_CALL(hipEventDestroy(start));
  CHECK_CUDA_CALL(hipEventDestroy(stop));

  return 0;
}