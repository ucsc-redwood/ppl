#include <spdlog/spdlog.h>

#include <algorithm>
#include <array>
#include <glm/glm.hpp>

#include "cuda/helper.cuh"
#include "cuda/kernels/01_morton.cuh"

struct Task {
  void allocate(const int n) {
    this->n = n;
    MALLOC_MANAGED(&u_input, n);
    MALLOC_MANAGED(&u_output, n);

    std::generate(u_input, u_input + n, []() {
      return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
    });
  }

  ~Task() {
    CUDA_FREE(u_input);
    CUDA_FREE(u_output);
  }

  int n;
  glm::vec4* u_input;
  unsigned int* u_output;
};

void execute(Task& t, hipStream_t* stream, const int tid) {
  CHECK_CUDA_CALL(
      hipStreamAttachMemAsync(stream[tid], t.u_input, 0, hipMemAttachSingle));

  gpu::k_ComputeMortonCode<<<1, 768, 0, stream[tid]>>>(
      t.u_input, t.u_output, t.n, 0.0f, 100.0f);
}

int main() {
  constexpr auto n = 1 << 20;  // 1M elements

  constexpr auto n_tasks = 100;

  constexpr auto n_streams = 4;
  std::array<hipStream_t, n_streams> streams;

  //   std::array<Task, n_streams> tasks{Task(n), Task(n), Task(n), Task(n)};

  std::vector<Task> tasks(n_tasks);  // to dos
  for (auto& task : tasks) {
    task.allocate(n);
  }

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  constexpr auto min_coord = 0.0f;
  constexpr auto range = 100.0f;

  hipEvent_t start, stop;
  CHECK_CUDA_CALL(hipEventCreate(&start));
  CHECK_CUDA_CALL(hipEventCreate(&stop));

  CHECK_CUDA_CALL(hipEventRecord(start, nullptr));

  // ------------------------------

  for (auto i = 0; i < n_tasks; ++i) {
    execute(tasks[i], streams.data(), i % n_streams);
  }

  // ------------------------------

  SYNC_DEVICE();

  CHECK_CUDA_CALL(hipEventRecord(stop, nullptr));
  CHECK_CUDA_CALL(hipEventSynchronize(stop));

  float milliseconds = 0;
  CHECK_CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  spdlog::info("Total time: {} ms", milliseconds);

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }

  CHECK_CUDA_CALL(hipEventDestroy(start));
  CHECK_CUDA_CALL(hipEventDestroy(stop));

  return 0;
}