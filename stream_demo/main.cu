#include "hip/hip_runtime.h"
#include <spdlog/spdlog.h>

#include <algorithm>
#include <array>
#include <glm/glm.hpp>

#include "cuda/helper.cuh"
#include "cuda/kernels/01_morton.cuh"

__global__ void k_DoSomethingA(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(sqrt(u_input[j].x));
  }
}

__global__ void k_DoSomethingB(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(pow(u_input[j].y, 6.66));
  }
}

__global__ void k_DoSomethingC(glm::vec4* u_input,
                               unsigned int* u_output,
                               const int n) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto j = i; j < n; j += stride) {
    u_output[j] = static_cast<unsigned int>(exp(u_input[j].z));
  }
}

struct Task {
  void allocate(const int n) {
    this->n = n;
    MALLOC_MANAGED(&u_input, n);
    MALLOC_MANAGED(&u_output, n);

    std::generate(u_input, u_input + n, []() {
      return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
    });
  }

  ~Task() {
    CUDA_FREE(u_input);
    CUDA_FREE(u_output);
  }

  int n;
  glm::vec4* u_input;
  unsigned int* u_output;
};

void execute(Task& t, hipStream_t* stream, const int tid) {
  CHECK_CUDA_CALL(
      hipStreamAttachMemAsync(stream[tid], t.u_input, 0, hipMemAttachSingle));

  if (tid == 0) {
    k_DoSomethingA<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 1) {
    k_DoSomethingB<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else if (tid == 2) {
    k_DoSomethingC<<<1, 512, 0, stream[tid]>>>(t.u_input, t.u_output, t.n);
  } else {
    gpu::k_ComputeMortonCode<<<1, 512, 0, stream[tid]>>>(
        t.u_input, t.u_output, t.n, 0.0f, 100.0f);
  }
}

int main() {
  constexpr auto n = 1 << 20;  // 1M elements

  constexpr auto n_tasks = 100;

  volatile auto h_original_input = new glm::vec4[n];
  std::generate(h_original_input, h_original_input + n, []() {
    return glm::vec4{1.0f, 2.0f, 3.0f, 4.0f};
  });

  constexpr auto n_streams = 4;
  std::array<hipStream_t, n_streams> streams;

  //   std::array<Task, n_streams> tasks{Task(n), Task(n), Task(n), Task(n)};

  //   std::vector<Task> tasks(n_tasks);  // to dos
  //   for (auto& task : tasks) {
  //     task.allocate(n);
  //   }

  std::array<Task, n_streams> tasks{Task(), Task(), Task(), Task()};

  for (auto& task : tasks) {
    task.allocate(n);
  }

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  constexpr auto min_coord = 0.0f;
  constexpr auto range = 100.0f;

  hipEvent_t start, stop;
  CHECK_CUDA_CALL(hipEventCreate(&start));
  CHECK_CUDA_CALL(hipEventCreate(&stop));

  CHECK_CUDA_CALL(hipEventRecord(start, nullptr));

  // ------------------------------

  for (auto i = 0; i < n_tasks; ++i) {
    const auto my_id = i % n_streams;
    std::copy_n(h_original_input, n, tasks[my_id].u_input);

    execute(tasks[my_id], streams.data(), my_id);
  }

  // ------------------------------

  SYNC_DEVICE();

  CHECK_CUDA_CALL(hipEventRecord(stop, nullptr));
  CHECK_CUDA_CALL(hipEventSynchronize(stop));

  float milliseconds = 0;
  CHECK_CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  spdlog::info("Total time: {} ms", milliseconds);

  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }

  CHECK_CUDA_CALL(hipEventDestroy(start));
  CHECK_CUDA_CALL(hipEventDestroy(stop));

  delete[] h_original_input;
  return 0;
}