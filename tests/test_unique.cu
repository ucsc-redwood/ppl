#include <gtest/gtest.h>

#include <hipcub/hipcub.hpp>
#include <numeric>
#include <random>
#include <vector>

#include "cuda/dispatchers/unique_dispatch.cuh"
#include "cuda/unified_vector.cuh"

static void Test_Unique(const int n, const int n_blocks) {
  cu::unified_vector<unsigned int> u_data(n);
  cu::unified_vector<unsigned int> u_output(n);

  std::mt19937 gen(114514);
  std::uniform_int_distribution<unsigned int> dis(0, n / 2);

  std::generate(
      u_data.begin(), u_data.end(), [&dis, &gen]() { return dis(gen); });
  std::sort(u_data.begin(), u_data.end());

  hipStream_t stream;
  CHECK_CUDA_CALL(hipStreamCreate(&stream));

  // temporary memory
  cu::unified_vector<int> u_flag_heads(n);
  const auto prefix_sum_num_tiles =
      hipcub::DivideAndRoundUp(n, gpu::PrefixSumAgent<int>::tile_size);
  cu::unified_vector<int> u_auxiliary(prefix_sum_num_tiles);

  gpu::dispatch_Unique(n_blocks,
                       stream,
                       u_data.data(),
                       u_output.data(),
                       u_flag_heads.data(),
                       u_auxiliary.data(),
                       n);
  SYNC_STREAM(stream);

  // check with cpu
  std::vector cpu_data(u_data.begin(), u_data.end());
  const auto it = std::unique(cpu_data.begin(), cpu_data.end());
  const auto cpu_num_unique = std::distance(cpu_data.begin(), it);

  const auto is_equal = std::equal(
      u_output.begin(), u_output.begin() + cpu_num_unique, cpu_data.begin());

  for (int i = 0; i < 2048; ++i) {
    std::cout << "[" << i << "] " << u_data[i] << " -\t" << u_output[i] << "\t"
              << cpu_data[i] << std::endl;
  }

  EXPECT_TRUE(is_equal);

  CHECK_CUDA_CALL(hipStreamDestroy(stream));
}

TEST(Test_UniqueRegular, Test_Unique) {
  EXPECT_NO_FATAL_FAILURE(Test_Unique(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_Unique(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_Unique(1 << 20, 4));  // 1048576
}

int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}