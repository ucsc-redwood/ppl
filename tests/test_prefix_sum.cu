#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <hipcub/hipcub.hpp>
#include <numeric>
#include <vector>

#include "cuda/dispatchers/prefix_sum_dispatch.cuh"
#include "cuda/unified_vector.cuh"

#define CREATE_STREAM  \
  hipStream_t stream; \
  CHECK_CUDA_CALL(hipStreamCreate(&stream));

#define DESCROY_STREAM CHECK_CUDA_CALL(hipStreamDestroy(stream));

// -----------------------------------------------------------------------------
// Test Local Prefix Sum
// -----------------------------------------------------------------------------

static void Test_LocalPrefixSum(const int n, const int n_blocks) {
  CREATE_STREAM;

  const cu::unified_vector<unsigned int> u_input(n, 1);
  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;
  constexpr auto tile_size = gpu::PrefixSumAgent<int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);

  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  gpu::k_PrefixSumLocal<<<n_blocks, n_threads, 0, stream>>>(
      u_input.data(), u_output.data(), n, u_auxiliary.data());
  SYNC_DEVICE();

  // Each tile should independently have a prefix sum
  for (auto i = 0; i < n_tiles; ++i) {
    const auto offset = i * tile_size;

    // locally should [0, 1, 2, ... ,tile_size - 1]
    for (auto j = 0; j < tile_size; ++j) {
      if (offset + j < n) {
        EXPECT_EQ(u_output[offset + j], j);
      }
    }
  }

  // For each tile, the auxiliary memory should be the size of a tile
  for (auto i = 0; i < n_tiles - 1; ++i) {
    EXPECT_EQ(u_auxiliary[i], tile_size);
  }
  // for last block, the number is the remainder
  const auto last_block_size = n - (n_tiles - 1) * tile_size;
  EXPECT_EQ(u_auxiliary[n_tiles - 1], last_block_size);

  DESCROY_STREAM;
}

TEST(Test_LocalPrefixSum, RegularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 20, 4));  // 1048576
}

TEST(Test_LocalPrefixSum, IrregularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(114514, 1));
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(640 * 480, 2));
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1920 * 1080, 4));
}

// -----------------------------------------------------------------------------
// Single Block Exclusive Scan
// -----------------------------------------------------------------------------

static void Test_SingleBlock(const int n) {
  CREATE_STREAM;

  const cu::unified_vector<unsigned int> u_input(n, 1);
  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;

  gpu::k_SingleBlockExclusiveScan<<<1, n_threads, 0, stream>>>(
      u_input.data(), u_output.data(), n);
  SYNC_DEVICE();

  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(u_output[i], i);
  }

  DESCROY_STREAM;
}

static void Test_SingleBlock_Iota(const int n) {
  CREATE_STREAM;

  cu::unified_vector<unsigned int> u_input(n);
  std::iota(u_input.begin(), u_input.end(), 0);

  cu::unified_vector<unsigned int> u_output(n);
  std::vector<unsigned int> cpu_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;

  gpu::k_SingleBlockExclusiveScan<<<1, n_threads, 0, stream>>>(
      u_input.data(), u_output.data(), n);
  SYNC_DEVICE();

  std::exclusive_scan(u_input.begin(), u_input.end(), cpu_output.begin(), 0);

  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(u_output[i], cpu_output[i]);
  }

  DESCROY_STREAM;
}

TEST(Test_SingleBlockExclusiveScan, RegularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 10));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 16));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 20));  // 1048576
}

TEST(Test_SingleBlockExclusiveScan, IrregularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(114514));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(640 * 480));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1920 * 1080));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(753413));
}

TEST(Test_SingleBlockExclusiveScan, RegularInput_Arbitary) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(1 << 10));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(1 << 16));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(1 << 20));  // 1048576
}

TEST(Test_SingleBlockExclusiveScan, IrregularInput_Arbitary) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(114514));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(640 * 480));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(1920 * 1080));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock_Iota(753413));
}

// -----------------------------------------------------------------------------
// Global Prefix Sum
// -----------------------------------------------------------------------------

static void Test_MakeGlobalSums(const int n, const int n_blocks) {
  CREATE_STREAM;

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;
  constexpr auto tile_size = gpu::PrefixSumAgent<int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);

  cu::unified_vector<unsigned int> u_local_sums(n);
  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  // This bug is very crazy.
  // don't put "u_auxiliary[tile_idx] = tile_size;" in this loop
  for (auto tile_idx = 0; tile_idx < n_tiles; ++tile_idx) {
    const auto offset = tile_idx * tile_size;
    std::iota(u_local_sums.begin() + offset,
              u_local_sums.begin() + offset + tile_size,
              0);
  }

  for (auto tile_idx = 0; tile_idx < n_tiles; ++tile_idx) {
    u_auxiliary[tile_idx] =
        u_local_sums[tile_idx * tile_size + tile_size - 1] + 1;
  }

  // perform prefix sum on u_auxiliary
  std::exclusive_scan(
      u_auxiliary.begin(), u_auxiliary.end(), u_auxiliary.begin(), 0);

  // -------

  cu::unified_vector<unsigned int> u_global_sums(n);

  gpu::k_MakeGlobalPrefixSum<<<n_blocks, n_threads, 0, stream>>>(
      u_local_sums.data(), u_auxiliary.data(), u_global_sums.data(), n);
  SYNC_DEVICE();

  for (auto i = 0; i < n; ++i) {
    const auto tile_idx = i / tile_size;

    EXPECT_EQ(u_global_sums[i], i);
  }

  DESCROY_STREAM;
}

TEST(Test_MakeGlobalSums, RegularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(1 << 20, 4));  // 1048576
}

TEST(Test_MakeGlobalSums, IrregularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(114514, 1));
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(640 * 480, 2));
  EXPECT_NO_FATAL_FAILURE(Test_MakeGlobalSums(1920 * 1080, 4));
}

// -----------------------------------------------------------------------------
// Global Prefix Sum
// -----------------------------------------------------------------------------

static void Test_PrefixSum(const int n, const int n_blocks) {
  CREATE_STREAM;

  const cu::unified_vector<unsigned int> u_data(n, 1);
  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto tile_size = gpu::PrefixSumAgent<unsigned int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);
  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  gpu::dispatch_PrefixSum(
      n_blocks, stream, u_data.data(), u_output.data(), u_auxiliary.data(), n);
  SYNC_STREAM(stream);

  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(u_output[i], i);
  }

  DESCROY_STREAM;
}

static void Test_PrefixSum_Iota(const int n, const int n_blocks) {
  CREATE_STREAM;

  cu::unified_vector<unsigned int> u_data(n);
  std::iota(u_data.begin(), u_data.end(), 0);

  cu::unified_vector<unsigned int> u_output(n);
  std::vector<unsigned int> cpu_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<unsigned int>::n_threads;
  constexpr auto tile_size = gpu::PrefixSumAgent<unsigned int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);
  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  gpu::k_PrefixSumLocal<<<n_blocks, n_threads, 0, stream>>>(
      u_data.data(), u_output.data(), n, u_auxiliary.data());
  SYNC_DEVICE();

  // std::vector<unsigned int> cpu_tmp(tile_size);
  // std::exclusive_scan(
  //     u_data.begin(), u_data.begin() + tile_size, cpu_tmp.begin(), 0);

  // for (auto i = 0; i < n; ++i) {
  //   std::cout << i << ":\t" << u_output[i];
  //   if (i < tile_size) {
  //     std::cout << " - " << cpu_tmp[i];
  //   }
  //   std::cout << '\n';
  // }

  for (auto tile_idx = 0; tile_idx < n_tiles; ++tile_idx) {
    std::cout << "u_aux[" << tile_idx << "]\t" << u_auxiliary[tile_idx]
              << "\t(tile size: " << tile_size << ")" << '\n';
  }

  gpu::k_SingleBlockExclusiveScan<<<1, n_threads, 0, stream>>>(
      u_auxiliary.data(), u_auxiliary.data(), n_tiles);
  SYNC_DEVICE();

  for (auto tile_idx = 0; tile_idx < n_tiles; ++tile_idx) {
    std::cout << "u_aux_summed[" << tile_idx << "]\t" << u_auxiliary[tile_idx]
              << "\t(tile size: " << tile_size << ")" << '\n';
  }

  gpu::k_MakeGlobalPrefixSum<<<n_blocks, n_threads, 0, stream>>>(
      u_output.data(), u_auxiliary.data(), u_output.data(), n);
  SYNC_DEVICE();

  // gpu::dispatch_PrefixSum(
  //     n_blocks, stream, u_data.data(), u_output.data(), u_auxiliary.data(),
  //     n);
  // SYNC_STREAM(stream);

  std::exclusive_scan(u_data.begin(), u_data.end(), cpu_output.begin(), 0);

  for (auto i = 0; i < n; ++i) {
    std::cout << "u_output[" << i << "]\t" << u_output[i]
              << "\tvs. cpu: " << cpu_output[i]
              << (u_output[i] == cpu_output[i] ? "" : " <--- ERROR!") << '\n';
  }

  // for (auto i = 0; i < 2 * tile_size; ++i) {
  //   EXPECT_EQ(u_output[i], cpu_output[i]);
  // }

  DESCROY_STREAM;
}

TEST(Test_PrefixSum, RegularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 20, 4));  // 1048576
}

TEST(Test_PrefixSum, IrregularInput) {
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(114514, 1));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(640 * 480, 8));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1920 * 1080, 16));
}

TEST(Test_PrefixSum, RegularInput_Arbitary) {
  // EXPECT_NO_FATAL_FAILURE(Test_PrefixSum_Iota(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum_Iota(1 << 16, 1));  // 65536
  // EXPECT_NO_FATAL_FAILURE(Test_PrefixSum_Iota(1 << 20, 4));  // 1048576
}

// TEST(Test_PrefixSum, Test_PrefixSumArbitaryInput_Int) {
//   EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 10, 1));
//   EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 16, 2));
//   EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 20, 4));
// }

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}