#include <gtest/gtest.h>

#include <hipcub/hipcub.hpp>
#include <numeric>
#include <vector>

#include "cuda/dispatchers/prefix_sum_dispatch.cuh"
#include "cuda/unified_vector.cuh"

#define CREATE_STREAM  \
  hipStream_t stream; \
  CHECK_CUDA_CALL(hipStreamCreate(&stream));

#define DESCROY_STREAM CHECK_CUDA_CALL(hipStreamDestroy(stream));

// -----------------------------------------------------------------------------
// Test Local Prefix Sum
// -----------------------------------------------------------------------------

static void Test_LocalPrefixSum(const int n, const int n_blocks) {
  CREATE_STREAM;

  const cu::unified_vector<unsigned int> u_input(n, 1);

  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;
  constexpr auto tile_size = gpu::PrefixSumAgent<int>::tile_size;

  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);

  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  gpu::k_PrefixSumLocal<<<n_blocks, n_threads, 0, stream>>>(
      u_input.data(), u_output.data(), n, u_auxiliary.data());
  SYNC_DEVICE();

  for (auto i = 0; i < n_tiles; ++i) {
    const auto offset = i * tile_size;
    for (auto j = 0; j < tile_size; ++j) {
      if (offset + j < n) {
        EXPECT_EQ(u_output[offset + j], j);
      }
    }
  }

  for (auto i = 0; i < n_tiles - 1; ++i) {
    // std::cout << "=========== " << i << "\t" << u_auxiliary[i] << std::endl;
    EXPECT_EQ(u_auxiliary[i], tile_size);
  }
  // // for last block, the number is something else
  // const auto last_block_size = n - (n_tiles - 1) * tile_size;
  // EXPECT_EQ(u_auxiliary[n_tiles - 1], last_block_size);

  DESCROY_STREAM;
}

TEST(LocalPrefixSum_Regular, Test_LocalPrefixSum) {
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1 << 20, 4));  // 1048576
}

TEST(LocalPrefixSum_Irregular, Test_LocalPrefixSum) {
  EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(114514, 1));
  // EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(640 * 480, 2));
  // EXPECT_NO_FATAL_FAILURE(Test_LocalPrefixSum(1920 * 1080, 4));
}

// -----------------------------------------------------------------------------
// Single Block Exclusive Scan
// -----------------------------------------------------------------------------

static void Test_SingleBlock(const int n) {
  CREATE_STREAM;

  const cu::unified_vector<unsigned int> u_input(n, 1);
  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto n_threads = gpu::PrefixSumAgent<int>::n_threads;

  gpu::k_SingleBlockExclusiveScan<<<1, n_threads, 0, stream>>>(
      u_input.data(), u_output.data(), n);
  SYNC_DEVICE();

  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(u_output[i], i);
  }

  DESCROY_STREAM;
}

TEST(SingleBlockExclusiveScan_Regular, Test_SingleBlock) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 10));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 16));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1 << 20));  // 1048576
}

TEST(SingleBlockExclusiveScan_Irregular, Test_SingleBlock) {
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(114514));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(640 * 480));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(1920 * 1080));
  EXPECT_NO_FATAL_FAILURE(Test_SingleBlock(753413));
}

// -----------------------------------------------------------------------------
// Global Prefix Sum
// -----------------------------------------------------------------------------

static void Test_PrefixSum(const int n, const int n_blocks) {
  const cu::unified_vector<unsigned int> u_data(n, 1);
  cu::unified_vector<unsigned int> u_output(n);

  constexpr auto tile_size = gpu::PrefixSumAgent<unsigned int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);
  cu::unified_vector<unsigned int> u_auxiliary(n_tiles);

  hipStream_t stream;
  CHECK_CUDA_CALL(hipStreamCreate(&stream));

  gpu::dispatch_PrefixSum(
      n_blocks, stream, u_data.data(), u_output.data(), u_auxiliary.data(), n);
  SYNC_STREAM(stream);

  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(u_output[i], i);
  }

  CHECK_CUDA_CALL(hipStreamDestroy(stream));
}

static void Test_PrefixSumArbitaryInput_Int(const int n, const int n_blocks) {
  cu::unified_vector<int> u_data(n);
  cu::unified_vector<int> u_output(n);

  std::iota(u_data.begin(), u_data.end(), 1);
  const std::vector cpu_backup_data(u_data.begin(), u_data.end());

  std::vector<int> cpu_output(n);
  std::exclusive_scan(u_data.begin(), u_data.end(), cpu_output.begin(), 0);

  constexpr auto tile_size = gpu::PrefixSumAgent<int>::tile_size;
  const auto n_tiles = hipcub::DivideAndRoundUp(n, tile_size);
  cu::unified_vector<int> u_auxiliary(n_tiles);

  hipStream_t stream;
  CHECK_CUDA_CALL(hipStreamCreate(&stream));

  gpu::dispatch_PrefixSum(
      n_blocks, stream, u_data.data(), u_output.data(), u_auxiliary.data(), n);
  SYNC_STREAM(stream);

  auto is_equal =
      std::equal(u_output.begin(), u_output.end(), cpu_output.begin());
  EXPECT_TRUE(is_equal);

  // Also check if u_data is modified (it should not)
  is_equal = std::equal(u_data.begin(), u_data.end(), cpu_backup_data.begin());
  EXPECT_TRUE(is_equal);

  CHECK_CUDA_CALL(hipStreamDestroy(stream));
}

TEST(GlobalPrefixSumRegular, Test_PrefixSum) {
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 10, 1));  // 1024
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 16, 2));  // 65536
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1 << 20, 4));  // 1048576
}

TEST(GlobalPrefixSumIrregular, Test_PrefixSum) {
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(114514, 1));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(640 * 480, 8));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSum(1920 * 1080, 16));
}

TEST(GlobalPrefixSumArbitraryInput, Test_PrefixSumArbitaryInput_Int) {
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 10, 1));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 16, 2));
  EXPECT_NO_FATAL_FAILURE(Test_PrefixSumArbitaryInput_Int(1 << 20, 4));
}

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}