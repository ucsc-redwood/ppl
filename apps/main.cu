#include <omp.h>
#include <spdlog/spdlog.h>

#include "app_params.hpp"
// #include "baselines.h"
#include "handlers/pipe.cuh"
#include "kernels_fwd.h"

void runAllStagesOnGpu(const AppParams& params,
                       const hipStream_t stream,
                       const std::unique_ptr<Pipe>& pipe) {
  gpu::v2::dispatch_Init(params.n_blocks, stream, *pipe);
  gpu::v2::dispatch_ComputeMorton(params.n_blocks, stream, *pipe);
  gpu::v2::dispatch_RadixSort(params.n_blocks, stream, pipe->sort);
  gpu::v2::dispatch_RemoveDuplicates(
      params.n_blocks, stream, pipe->sort.data(), pipe->unique);
  SYNC_STREAM(stream);

  const auto n_unique = pipe->unique.attemptGetNumUnique();

  gpu::v2::dispatch_BuildRadixTree(
      params.n_blocks, stream, pipe->unique.begin(), n_unique, pipe->brt);

  SYNC_STREAM(stream);
  // peek 10 brt nodes
  for (auto i = 0; i < 10; ++i) {
    spdlog::info("BRT node {}: {}", i, pipe->brt.u_prefix_n[i]);
  }

  spdlog::info("Unique keys: {}/{} ({}%)",
               n_unique,
               pipe->n,
               100.0 * n_unique / pipe->n);

  // auto is_sorted = std::is_sorted(pipe->sort.begin(), pipe->sort.end());
  // spdlog::info("Is sorted (after): {}", is_sorted);
}

int main(const int argc, const char** argv) {
  AppParams params(argc, argv);
  params.print_params();

  switch (params.log_level) {
    case 0:
      spdlog::set_level(spdlog::level::off);
      break;
    case 1:
      spdlog::set_level(spdlog::level::info);
      break;
    case 2:
      spdlog::set_level(spdlog::level::debug);
      break;
    case 3:
      spdlog::set_level(spdlog::level::trace);
      break;
    default:
      spdlog::set_level(spdlog::level::info);
      break;
  }

  omp_set_num_threads(params.n_threads);
#pragma omp parallel
  { spdlog::debug("Hello from thread {}", omp_get_thread_num()); }

  // ------------------------------
  constexpr auto n_streams = 1;
  const auto n_iterations = params.n_iterations;

  std::array<hipStream_t, n_streams> streams;
  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  const auto pipe = std::make_unique<Pipe>(
      params.n, params.min_coord, params.getRange(), params.seed);
  pipe->attachStreamGlobal(streams[0]);

  for (auto i = 0; i < n_iterations; ++i) {
    ++pipe->seed;
    runAllStagesOnGpu(params, streams[0], pipe);
  }

  // ------------------------------

  spdlog::info("Done");
  for (const auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }
  return 0;
}