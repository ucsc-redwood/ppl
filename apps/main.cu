#include <omp.h>
#include <spdlog/spdlog.h>

#include "app_params.hpp"
#include "handlers/pipe.cuh"
#include "kernels_fwd.h"

void runAllStagesOnGpu(const AppParams& params,
                       const hipStream_t stream,
                       const std::unique_ptr<Pipe>& pipe) {
  gpu::v2::dispatch_Init(params.n_blocks, stream, *pipe);
  gpu::v2::dispatch_ComputeMorton(params.n_blocks, stream, *pipe);
  gpu::v2::dispatch_RadixSort(params.n_blocks, stream, pipe->sort);
  gpu::v2::dispatch_RemoveDuplicates(
      params.n_blocks, stream, pipe->sort.data(), pipe->unique);

  SYNC_STREAM(stream);
  const auto n_unique = pipe->unique.attemptGetNumUnique();
  pipe->brt.setNumBrtNodes(n_unique - 1);

  gpu::v2::dispatch_BuildRadixTree(
      params.n_blocks, stream, pipe->unique.begin(), n_unique, pipe->brt);
  gpu::v2::dispatch_EdgeCount(
      params.n_blocks, stream, pipe->brt, pipe->u_edge_count);
  gpu::v2::dispatch_EdgeOffset_safe(params.n_blocks,
                                    stream,
                                    pipe->u_edge_count,
                                    pipe->u_edge_offset,
                                    pipe->brt.getNumBrtNodes());

  SYNC_STREAM(stream);
  // const auto n_unique = pipe->attemptGetNumOctNodes();
  const auto n_oct_nodes = pipe->u_edge_offset[pipe->brt.getNumBrtNodes() - 1];

  gpu::v2::dispatch_BuildOctree(params.n_blocks,
                                stream,
                                pipe->brt,
                                pipe->sort.data(),
                                pipe->u_edge_offset,
                                pipe->u_edge_count,
                                pipe->oct,
                                params.min_coord,
                                params.getRange());

  gpu::v2::dispatch_LinkOctreeNodes(params.n_blocks,
                                    stream,
                                    pipe->u_edge_offset,
                                    pipe->u_edge_count,
                                    pipe->sort.data(),
                                    pipe->brt,
                                    pipe->oct);

  SYNC_STREAM(stream);

  // // peek 10 oct nodes
  // for (auto i = 0; i < 10; ++i) {
  //   spdlog::trace("oct node[{}]: {}", i, pipe->oct.u_children[i][0]);
  // }

  // spdlog::info("Unique keys: {}/{} ({}%)",
  //              n_unique,
  //              pipe->n,
  //              100.0 * n_unique / pipe->n);
  // spdlog::info("Oct nodes: {}/{} ({}%)",
  //              n_oct_nodes,
  //              pipe->n,
  //              100.0 * n_oct_nodes / pipe->n);

  // merge the two spdlog calls, then set precision to 2 decimal places

  spdlog::info("Unique keys: {} / {} ({}%) | Oct nodes: {} / {} ({}%)",
               n_unique,
               pipe->n,
               100.0f * n_unique / pipe->n,
               n_oct_nodes,
               pipe->n,
               100.0f * n_oct_nodes / pipe->n);
}

int main(const int argc, const char** argv) {
  AppParams params(argc, argv);
  params.print_params();

  switch (params.log_level) {
    case 0:
      spdlog::set_level(spdlog::level::off);
      break;
    case 1:
      spdlog::set_level(spdlog::level::info);
      break;
    case 2:
      spdlog::set_level(spdlog::level::debug);
      break;
    case 3:
      spdlog::set_level(spdlog::level::trace);
      break;
    default:
      spdlog::set_level(spdlog::level::info);
      break;
  }

  omp_set_num_threads(params.n_threads);
#pragma omp parallel
  { spdlog::debug("Hello from thread {}", omp_get_thread_num()); }

  // ------------------------------
  constexpr auto n_streams = 1;
  const auto n_iterations = params.n_iterations;

  std::array<hipStream_t, n_streams> streams;
  for (auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamCreate(&stream));
  }

  const auto pipe = std::make_unique<Pipe>(
      params.n, params.min_coord, params.getRange(), params.seed);
  pipe->attachStreamGlobal(streams[0]);

  for (auto i = 0; i < n_iterations; ++i) {
    ++pipe->seed;
    runAllStagesOnGpu(params, streams[0], pipe);
  }

  // ------------------------------

  spdlog::info("Done");
  for (const auto& stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }
  return 0;
}